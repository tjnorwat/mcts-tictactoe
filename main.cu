
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to print "Hello, World!" from the GPU
__global__ void helloKernel() {
    printf("Hello, World! from GPU thread %d\n", threadIdx.x);
}

int main() {
    // Print "Hello, World!" from the CPU
    printf("Hello, World! from CPU\n");
    
    // Launch the kernel with 5 threads
    helloKernel<<<1, 5>>>();
    
    // Wait for the GPU to finish
    hipDeviceSynchronize();
    
    return 0;
}
